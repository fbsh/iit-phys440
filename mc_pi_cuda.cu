
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>


__global__ void trial(int seed, bool count_d[], double x_d[], double y_d[]) {
    long long id = blockIdx.x * blockDim.x + threadIdx.x;
    double x = x_d[id], y = y_d[id];
    if(x*x + y*y <= 1) {
        count_d[id] = true;
    }
    else {
        count_d[id] = false;
    }
}

int main(int argc, char* argv[]) {
    int seed = time(NULL);
    long long total = 1e6;  // Default 1 million samples
    int tn = 128;             // Default 128 threads

    if(argc >= 2) {
        total = atoi(argv[1]);  // Get the number of samples from the arg
    }
    if(argc >= 3) {
        tn = atoi(argv[2]);     // Get the number of threads from the arg
    }
    dim3 threads(tn);
    dim3 blocks((total+tn-1) / tn);
    long long real_total = threads.x * blocks.x;

    bool* count_h = new bool[real_total];
    bool* count_d;
    double* x_h = new double[real_total];
    double* y_h = new double[real_total];
    double* x_d, *y_d;
    for(long long i = 0; i < real_total; i++) {
        x_h[i] = (double)rand() / RAND_MAX;
        y_h[i] = (double)rand() / RAND_MAX;
    }
    hipMalloc(&count_d, real_total * sizeof(bool));  // Graphic memory for saving results
    hipMalloc(&x_d, real_total * sizeof(double));    // random number array x
    hipMalloc(&y_d, real_total * sizeof(double));    // random number array y
    hipMemcpy(x_d, x_h, real_total * sizeof(double), hipMemcpyHostToDevice);  // copy random number array
    hipMemcpy(y_d, y_h, real_total * sizeof(double), hipMemcpyHostToDevice);  // copy random number array

    trial<<<blocks, threads>>>(seed, count_d, x_d, y_d);

    hipMemcpy(count_h, count_d, real_total * sizeof(bool), hipMemcpyDeviceToHost);

    long long count = 0;
    for(long long i = 0; i < real_total; i++) {
        if(count_h[i]) {
            count++;
        }
    }
    double pi = 4 * (double)count / real_total;
    double r = 1;
    double volume = 4 * (double)pi * r * r * r / 3;
    double surface = 4 * (double)pi * r * r;
    double bound_volume = r * r * r;
    double bound_surface =  6 * r * r;

    printf("[+] total                       = %lld\n", real_total);  // The actual total may be different from the parameter, depending on whether it is divisible or not
    printf("[+] count                       = %lld\n", count);
    printf("[+] pi                          = %f\n", pi);
    printf("[+] loss                        = %e\n", acos(-1) - pi);

    printf("[+] volume of the unit sphere   = %f\n", volume);
    printf("[+] surface of the unit sphere  = %f\n", surface);
    printf("[+] volume of the box           = %f\n", bound_volume);
    printf("[+] surface of the box          = %f\n", bound_surface);

    printf("\nBlocks  = %d\n", blocks.x);
    printf("Threads = %d\n", threads.x);

    return 0;
}